#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <time.h>
#include <sys/time.h>

// includes, kernels
#include "vector_dot_product_kernel.cu"

void run_test(unsigned int);
float compute_on_device(float *, float *,int);
void check_for_error(char *);
extern "C" float compute_gold( float *, float *, unsigned int);

int 
main( int argc, char** argv) 
{
	if(argc != 2){
		printf("Usage: vector_dot_product <num elements> \n");
		exit(0);	
	}
	unsigned int num_elements = atoi(argv[1]);
	run_test(num_elements);
	return 0;
}

void 
run_test(unsigned int num_elements) 
{
	// Obtain the vector length
	unsigned int size = sizeof(float) * num_elements;

	// Allocate memory on the CPU for the input vectors A and B
	float *A = (float *)malloc(size);
	float *B = (float *)malloc(size);
	
	// Randomly generate input data. Initialize the input data to be floating point values between [-.5 , 5]
	printf("Generating random vectors with values between [-.5, .5]. \n");	
	srand(time(NULL));
	for(unsigned int i = 0; i < num_elements; i++){
		A[i] = (float)rand()/(float)RAND_MAX - 0.5;
		B[i] = (float)rand()/(float)RAND_MAX - 0.5;
		//printf("A = %d;  B = %d. \n", A[i], B[i]);
	}
	
	printf("Generating dot product on the CPU. \n");
	struct timeval start, stop;	
	gettimeofday(&start, NULL);
	
	float reference = compute_gold(A, B, num_elements);
    
	gettimeofday(&stop, NULL);
	printf("Execution time on CPU = %fs. \n", (float)(stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000));
	
	/* Edit this function to compute the result vector on the GPU. 
       The result should be placed in the gpu_result variable. */
	float gpu_result = compute_on_device(A, B, num_elements);

	printf("Result on CPU: %f, result on GPU: %f. \n", reference, gpu_result);
    printf("Epsilon: %f. \n", fabsf(reference - gpu_result));

	// cleanup memory
	free(A);
	free(B);
	
	return;
}

/* Edit this function to compute the dot product on the device using atomic intrinsics. */
float 
compute_on_device(float *A_on_host, float *B_on_host, int num_elements)
{
	float *A_on_device = NULL;
    float *B_on_device = NULL;
	float *C_on_device = NULL;
	
    hipMalloc ((void **) &A_on_device, num_elements * sizeof (float));
    hipMemcpy (A_on_device, A_on_host, num_elements * sizeof (float), hipMemcpyHostToDevice);
	
    hipMalloc ((void **) &B_on_device, num_elements * sizeof (float));
    hipMemcpy (B_on_device, B_on_host, num_elements * sizeof (float), hipMemcpyHostToDevice);
	
	hipMalloc ((void **) &C_on_device, num_elements * sizeof (float));
	hipMemset( C_on_device, 0.0f, GRID_SIZE * sizeof(float) );
	
	int *mutex = NULL;
    hipMalloc((void **)&mutex, sizeof(int));
    hipMemset(mutex, 0, sizeof(int));
		
    dim3 thread_block (THREAD_BLOCK_SIZE, 1, 1);
    dim3 grid (GRID_SIZE, 1);
   
    struct timeval start, stop;	
	gettimeofday(&start, NULL);
	
    vector_dot_product_kernel <<< grid, thread_block >>> (A_on_device, B_on_device, C_on_device, num_elements, mutex);	 
	hipDeviceSynchronize();
	
	gettimeofday(&stop, NULL);
	printf("Execution time on GPU = %fs. \n", (float)(stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000));
	
	float gpu_result = 0.0f;
    hipMemcpy(&gpu_result, C_on_device, sizeof(float), hipMemcpyDeviceToHost);
    
	hipFree(A_on_device); 
	hipFree(B_on_device);
	hipFree(C_on_device);
	
	return gpu_result;  
}
 
// This function checks for errors returned by the CUDA run time
void 
check_for_error(char *msg)
{
	hipError_t err = hipGetLastError();
	if(hipSuccess != err){
		printf("CUDA ERROR: %s (%s). \n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
} 
